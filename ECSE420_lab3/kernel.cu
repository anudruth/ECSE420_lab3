
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

#define n 0.0002    //given parameter
#define p 0.5		//given parameter
#define G 0.75		//given parameter

#define N 4			//size

int seqSolver(float u[][N], float u1[][N], float u2[][N]);
int display(float u[][N]);


int main(int argc, char* argv[])
{
	int iterations = atoi(argv[1]);

	float seqDrum_U[N][N] = { 0 };
	float seqDrum_U1[N][N] = { 0 };
	float seqDrum_U2[N][N] = { 0 };

	seqDrum_U1[N / 2][N / 2] += 1.0f;

	for (int i = 0; i < iterations; i++) {
		seqSolver(seqDrum_U, seqDrum_U1, seqDrum_U2);
		printf("U[N/2][N/2] after %d interation: %3.6f\n", i, seqDrum_U[N / 2][N / 2]);
		display(seqDrum_U);
		memcpy(seqDrum_U2, seqDrum_U1, N * N * sizeof(float));
		memcpy(seqDrum_U1, seqDrum_U, N * N * sizeof(float));
	}
	

	return 0;
}

int seqSolver(float u[][N], float u1[][N], float u2[][N]) {
	
	for (int i = 1; i < N - 1; i++) {
		for (int j = 1; j < N - 1; j++) {
			u[i][j] = (p*(u1[i-1][j]+u1[i+1][j]+u1[i][j-1]+u1[i][j+1]-4*u1[i][j])+2*u1[i][j]-(1-n)*u2[i][j]) / (1 + n);
		}
	}

	for (int i = 1; i < N - 1; i++) {
		u[0][i] = G * u[1][i];
		u[N - 1][i] = G * u[N - 2][i];
		u[i][0] = G * u[i][1];
		u[i][N - 1] = G * u[i][N - 2];
	}

	u[0][0] = G * u[1][0];
	u[N - 1][0] = G * u[N - 2][0];
	u[0][N - 1] = G * u[0][N - 2];
	u[N - 1][N - 1] = G * u[N - 1][N - 2];

	return 0;
}


int display(float u[][N]) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("(%d,%d): %3.6f ", i, j, u[i][j]);
		}
		printf("\n");
	}
	printf("\n");

	return 0;
}

