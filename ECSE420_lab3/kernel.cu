#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

#define n 0.0002    //given parameter
#define p 0.5		//given parameter
#define G 0.75		//given parameter

#define N 4			//size
#define M 512		//size part 3

int seqSolver(float u[][N], float u1[][N], float u2[][N]);
int display(float u[][N]);
int display(float* u);

__global__ void parallelSolverP1(float* u, float* u1, float* u2)
{
	if ((threadIdx.x > N) && (threadIdx.x < (N * N - N)) && (threadIdx.x % N != 0) && (threadIdx.x % N + 1 != N)) {
		u[threadIdx.x] = (p * (u1[threadIdx.x - N] + u1[threadIdx.x + N] + u1[threadIdx.x - 1] + u1[threadIdx.x + 1] - 4 * u1[threadIdx.x]) + 2 * u1[threadIdx.x] - (1 - n) * u2[threadIdx.x]) / (1 + n);
	}
}

__global__ void parallelSolverP2(float* u) 
{
	if (!((threadIdx.x > N) && (threadIdx.x < (N * N - N)) && (threadIdx.x % N != 0) && (threadIdx.x % N + 1 != N)) &&
		((threadIdx.x != 0) || (threadIdx.x != N * N - 1) || (threadIdx.x != N * N - N) || (threadIdx.x != N - 1))) 
	{
		if (threadIdx.x < N) u[threadIdx.x] = G * u[threadIdx.x + N];
		if ((threadIdx.x > N*N-N) && (threadIdx.x < N*N)) u[threadIdx.x] = G * u[threadIdx.x-N];
		if (threadIdx.x % N == 0) u[threadIdx.x] = G * u[threadIdx.x + 1];
		if (threadIdx.x % N + 1 == N) u[threadIdx.x] = G * u[threadIdx.x - 1];
	}
}

__global__ void parallelSolverP3(float* u) 
{
	if (threadIdx.x == 0) u[threadIdx.x] = G * u[threadIdx.x + N];
	if (threadIdx.x == N * N - 1) u[threadIdx.x] = G * u[threadIdx.x - 1];
	if (threadIdx.x == N * N - N) u[threadIdx.x] = G * u[threadIdx.x - N];
	if (threadIdx.x == N - 1) u[threadIdx.x] = G * u[threadIdx.x - 1];
}

__global__ void parallelSolverP4(float* u, float* u1, float* u2)
{
	for (int index = (blockIdx.x * 1024 + threadIdx.x); index < M * M; index += 1024) {
		if ((index > M) && (index < (M * M - M)) && (index % M != 0) && (index % M + 1 != M)) {
			u[index] = (p * (u1[index - M] + u1[index + M] + u1[index - 1] + u1[index + 1] - 4 * u1[index]) + 2 * u1[index] - (1 - n) * u2[index]) / (1 + n);
		}
	}
}

__global__ void parallelSolverP5(float* u)
{
	for (int index = (blockIdx.x * 1024 + threadIdx.x); index < M * M; index += 1024) {
		if (!((index > M) && (index < (M * M - M)) && (index % M != 0) && (index % M + 1 != M)) &&
			((index != 0) || (index != M * M - 1) || (index != M * M - M) || (index != M - 1)))
		{
			if (index < M) u[index] = G * u[index + M];
			if ((index > M * M - M) && (index < M * M)) u[index] = G * u[index - M];
			if (index % M == 0) u[index] = G * u[index + 1];
			if (index % M + 1 == M) u[index] = G * u[index - 1];
		}
	}
}

__global__ void parallelSolverP6(float* u)
{
	for (int index = (blockIdx.x * 1024 + threadIdx.x); index < M * M; index += 1024) {
		if (index == 0) u[index] = G * u[index + M];
		if (index == M * M - 1) u[index] = G * u[index - 1];
		if (index == M * M - M) u[index] = G * u[index - M];
		if (index == M - 1) u[index] = G * u[index - 1];
	}
}

int main(int argc, char* argv[])
{
	int iterations = atoi(argv[1]);

	// sequential implimentation
	float seqDrum_U[N][N] = { 0 };
	float seqDrum_U1[N][N] = { 0 };
	float seqDrum_U2[N][N] = { 0 };

	seqDrum_U1[N / 2][N / 2] += 1.0f;


	printf("\nSequential implementation (part 1):\n");
	for (int i = 0; i < iterations; i++) {
		seqSolver(seqDrum_U, seqDrum_U1, seqDrum_U2);
		printf("U[N/2][N/2] after %d interation: %3.6f\n", i, seqDrum_U[N / 2][N / 2]);
		display(seqDrum_U);
		memcpy(seqDrum_U2, seqDrum_U1, N * N * sizeof(float));
		memcpy(seqDrum_U1, seqDrum_U, N * N * sizeof(float));
	}

	//free(seqDrum_U);
	//free(seqDrum_U1);
	//free(seqDrum_U2);

	//parallel implementation 
	float parDrum_U[N * N] = { 0 };
	float parDrum_U1[N * N] = { 0 };
	float parDrum_U2[N * N] = { 0 };

	float* d_parDrum_U;
	float* d_parDrum_U1;
	float* d_parDrum_U2;

	parDrum_U1[10] += 1;

	hipMallocManaged((void**)&d_parDrum_U, N * N * sizeof(float));
	hipMallocManaged((void**)&d_parDrum_U1, N * N * sizeof(float));
	hipMallocManaged((void**)&d_parDrum_U2, N * N * sizeof(float));

	hipMemcpy(d_parDrum_U, parDrum_U, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_parDrum_U1, parDrum_U1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_parDrum_U2, parDrum_U2, N * N * sizeof(float), hipMemcpyHostToDevice);

	printf("\nParallel implementation (part 2):\n");
	for (int i = 0; i < iterations; i++) {
		parallelSolverP1 << <1, 16 >> > (d_parDrum_U, d_parDrum_U1, d_parDrum_U2);
		hipDeviceSynchronize();
		parallelSolverP2 << <1, 16 >> > (d_parDrum_U);
		hipDeviceSynchronize();
		parallelSolverP3 << <1, 16 >> > (d_parDrum_U);
		hipDeviceSynchronize();
		printf("U[N/2][N/2] after %d interation: %3.6f\n", i, d_parDrum_U[10]);
		display(d_parDrum_U);
		hipMemcpy(d_parDrum_U2, d_parDrum_U1, N * N * sizeof(float), hipMemcpyDeviceToDevice);
		hipMemcpy(d_parDrum_U1, d_parDrum_U, N * N * sizeof(float), hipMemcpyDeviceToDevice);
	}
	/*
	free(parDrum_U);
	free(parDrum_U1);
	free(parDrum_U2);
	hipFree(d_parDrum_U);
	hipFree(d_parDrum_U1);
	hipFree(d_parDrum_U2);
	*/

	int iterations3 = 12;

	//parallel implementation 
	float* parDrum_U3 = (float*)malloc(M * M * sizeof(float));
	for (int i = 0; i < M * M; i++) {
		parDrum_U3[i] = 0;
	}

	int middle = (M / 2) * M + (M / 2);

	hipMallocManaged((void**)& d_parDrum_U, M * M * sizeof(float));
	hipMallocManaged((void**)& d_parDrum_U1, M * M * sizeof(float));
	hipMallocManaged((void**)& d_parDrum_U2, M * M * sizeof(float));

	hipMemcpy(d_parDrum_U, parDrum_U3, M * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_parDrum_U2, parDrum_U3, M * M * sizeof(float), hipMemcpyHostToDevice);
	parDrum_U3[middle] += 1;
	hipMemcpy(d_parDrum_U1, parDrum_U3, M * M * sizeof(float), hipMemcpyHostToDevice);

	printf("\nParallel implementation (part 3):\n");
	for (int i = 0; i < iterations3; i++) {
		parallelSolverP4 << <16, 1024 >> > (d_parDrum_U, d_parDrum_U1, d_parDrum_U2);
		hipDeviceSynchronize();
		parallelSolverP5 << <16, 1024 >> > (d_parDrum_U);
		hipDeviceSynchronize();
		parallelSolverP6 << <16, 1024 >> > (d_parDrum_U);
		hipDeviceSynchronize();
		printf("(256, 256) after %d interation: %3.6f\n", i, d_parDrum_U[middle]);
		hipMemcpy(d_parDrum_U2, d_parDrum_U1, M * M * sizeof(float), hipMemcpyDeviceToDevice);
		hipMemcpy(d_parDrum_U1, d_parDrum_U, M * M * sizeof(float), hipMemcpyDeviceToDevice);
	}

	free(parDrum_U3);

	return 0;
}

int seqSolver(float u[][N], float u1[][N], float u2[][N]) {
	
	for (int i = 1; i < N - 1; i++) {
		for (int j = 1; j < N - 1; j++) {
			u[i][j] = (p*(u1[i-1][j]+u1[i+1][j]+u1[i][j-1]+u1[i][j+1]-4*u1[i][j])+2*u1[i][j]-(1-n)*u2[i][j]) / (1 + n);
		}
	}

	for (int i = 1; i < N - 1; i++) {
		u[0][i] = G * u[1][i];
		u[N - 1][i] = G * u[N - 2][i];
		u[i][0] = G * u[i][1];
		u[i][N - 1] = G * u[i][N - 2];
	}

	u[0][0] = G * u[1][0];
	u[N - 1][0] = G * u[N - 2][0];
	u[0][N - 1] = G * u[0][N - 2];
	u[N - 1][N - 1] = G * u[N - 1][N - 2];

	return 0;
}


int display(float u[][N]) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("(%d,%d): %3.6f ", i, j, u[i][j]);
		}
		printf("\n");
	}
	printf("\n");

	return 0;
}

int display(float* u) {
	int index = 0;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("(%d,%d): %3.6f ", i, j, u[index++]);
		}
		printf("\n");
	}
	printf("\n");
	return 0;
}

